#include <assert.h>
#include <errno.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <sys/time.h>

#define DEBUG 1
#define NGRID 512
#define NBLOCK 65535
#define CUDA_CHECK(cmd) {hipError_t error = cmd; if(error!=hipSuccess){printf("<%s>:%i ",__FILE__,__LINE__); printf("[CUDA] Error: %s\n", hipGetErrorString(error));}}


/* The exponent given here determines the steps taken in the adding kernel. An
 * exponent of 1 results in rounding the size to 2^1 = 2, therefore, in every
 * step, two input fields are added and the size shrinks to half of what it was
 * before. This influences the size of the result buffer as well (the greater
 * this exponent is, the smaller the result will be). */
#define BASE_EXP 4
#define BASE (1 << BASE_EXP)

/* Define this to actually use host memory instead of copying the buffer to the
 * GPU (as it turns out, this may actually be worth it) */
#define USE_HOST_PTR


#ifdef USE_HOST_PTR
#define HOST_PTR_POLICY CL_MEM_USE_HOST_PTR
#else
#define HOST_PTR_POLICY CL_MEM_COPY_HOST_PTR
#endif

/**
 * These two functions provide std::chrono functionality (see cpp-stuff.cpp for
 * an explanation why they're extern).
 CUDA nvcc don't support it.
extern void clock_start(void);
extern long clock_delta(void);
*/

__global__ void k_iadd(unsigned *dest, char *sequence, unsigned seq_length)
{
    for (unsigned id =  blockIdx.x*blockDim.x+threadIdx.x; 
        id < seq_length; 
        id += blockDim.x*gridDim.x)
    {
    
        unsigned result = 0;
        unsigned in_start = id << BASE_EXP;

        if (in_start < seq_length)
        {
            for (unsigned i = in_start; i < in_start + BASE; i++)
            {
                char nucleobase = sequence[i];
                result += nucleobase != '-';
            }
        }

        dest[id] = result;
    }
}

__global__ void k_cadd(unsigned *buffer, unsigned doff, unsigned soff)
{
    unsigned id = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned in_start = soff + (id << BASE_EXP);
    unsigned out_pos = doff + id;
    unsigned result = 0;
    
    for (unsigned i = in_start; i < in_start + BASE; i++)
    {
        unsigned value = buffer[i];
        result += value;
    }
    
    buffer[out_pos] = result;
}

/**
 * Rounds a value x up to the next power of 2^exp.
 */
static long round_up_to_power_of_two(long x, int exp)
{
    assert(x > 0);

    x--;

    int i;
    for (i = 0; x; i++)
        x >>= exp;
    for (x = 1; i; i--)
        x <<= exp;

    return x;
}


/**
 * Loads a text file and returns a buffer with the contents.
 */
static char *load_text(const char *filename, long *length_ptr)
{
    FILE *fp = fopen(filename, "r");
    if (!fp)
    {
        fprintf(stderr, "Could not load file \"%s\": %s\n", filename, strerror(errno));
        return NULL;
    }

    fseek(fp, 0, SEEK_END);
    long length = ftell(fp);
    rewind(fp);

    long mem_len = length + 1;

    if (length_ptr)
        *length_ptr = mem_len;

    char *content = (char *)calloc(mem_len, 1);
    fread(content, 1, length, fp);
    fclose(fp);

    return content;
}


int main(int argc, char *argv[])
{
    dim3 grid1d(NGRID,1,1); 
    dim3 block1d(NBLOCK,1,1);
    unsigned clstsi, clstc, clsto, local_index;
    long delta_time;
    struct timeval start_time, end_time;

    if (argc < 2)
    {
        fprintf(stderr, "Usage: transalign_killer [--cldev=x.y] <input file>\n");
        fprintf(stderr, "  --cldev=x.y: x specifies the platform index, y the device index.\n");
        return 1;
    }


    long seq_length;
    //CUDA kernel input
    char *sequence = load_text(argv[argc - 1], &seq_length);

    if (!sequence)
        return 1;
//DEBUG
    for (int i=0; i<seq_length; ++i)
    {
        printf("%c", sequence[i]);
    }
    printf("\n");


    seq_length--; // Cut final 0 byte

    // FIXME: All the following code relies on seq_length being a multiple of BASE.

    long round_seq_length = round_up_to_power_of_two(seq_length, BASE_EXP);

    long res_length = 0;
    for (long len = round_seq_length / BASE; len; len /= BASE)
        res_length += len;

    printf("res_length: %d\n", res_length);

    // Use some random index to be searched for here
    unsigned letter_index = seq_length / 2;

    // Create the result buffer
    // CUDA kernel output
    unsigned *result = (unsigned *)malloc(res_length * sizeof(unsigned));
    unsigned *result_gpu;
    char *seq_gpu;

    //replace clock_start(); with gettimeofday()
    gettimeofday(&start_time, NULL);

#if DEBUG
    printf("GPU part started\n");
#endif
    /*** START OF ROCKET SCIENCE LEVEL RUNTIME-TIME INTENSIVE STUFF ***/

    // Bandwidth intensive stuff goes here
    // Copy the sequence to the video memory (or, generally speaking, the OpenCL device)
    CUDA_CHECK(hipMalloc((void**)&result_gpu, res_length * sizeof(unsigned)));//result_gpu
    CUDA_CHECK(hipMalloc((void**)&seq_gpu, seq_length*sizeof(char)));//seq_gpu
    CUDA_CHECK(hipMemcpy(seq_gpu, sequence, res_length * sizeof(char), hipMemcpyHostToDevice));

#if DEBUG
    printf("GPU malloc and cpy finised\n");
#endif

    //replace clock_delta(); with gettimeofday()
    gettimeofday(&end_time, NULL);
    long bw1_time = (end_time.tv_sec*1000000+end_time.tv_usec) - (start_time.tv_sec*1000000+start_time.tv_usec);

    // GPU intensive stuff goes here

    /**
     * First, transform every - and \0 into a 0 and every other character into a
     * 1. Then, add consecutive fields (BASE fields) together and store them at
     * the beginning of the result buffer.
     */
    
    //TODO: ADD correct kernel parameters
#if DEBUG
    printf("k_iadd launching\n");
#endif

    k_iadd<<<grid1d,block1d>>>(result_gpu, seq_gpu, seq_length);

#if DEBUG
    printf("k_iadd finished\n");
#endif

    CUDA_CHECK(hipMemcpy(result, result_gpu, res_length * sizeof(unsigned), hipMemcpyDeviceToHost));
#if DEBUG
    printf("result back\n");
    for (int i = 0; i < res_length; i++)
    {
        printf("%d ", result[i]);
    }
    printf("\n");
#endif

#if DEBUG
    printf("k_iadd result back\n");
#endif
    unsigned input_offset = 0, output_offset = round_seq_length / BASE;

    CUDA_CHECK(hipMemcpy(result_gpu, result, res_length * sizeof(unsigned), hipMemcpyHostToDevice));
#if DEBUG
    printf("k_cadd loop start\n");
#endif
    for (unsigned kernels = round_seq_length / (BASE * BASE); kernels > 0; kernels /= BASE)
    {
        /**
         * Then, do this addition recursively until there is only one kernel
         * remaining which calculates the total number of non-'-' and non-'\0'
         * characters.
         */
        //TODO: ADD correct kernel parameters
#if DEBUG
    printf("k_cadd loop %d\n", kernels);
#endif
        k_cadd<<<grid1d,block1d>>>(result_gpu, output_offset, input_offset);
        
        input_offset = output_offset;
        output_offset += kernels;
    }
 #if DEBUG
    printf("k_cadd loop end\n");
#endif   
    // Retrieve the result buffer 

 #if DEBUG
    printf("k_cadd loop end\n");
#endif   
    CUDA_CHECK(hipMemcpy(result, result_gpu, res_length * sizeof(unsigned), hipMemcpyDeviceToHost));
 #if DEBUG
    printf("k_cadd loop end\n");
#endif

    gettimeofday(&end_time, NULL);
    long gpu_time = (end_time.tv_sec*1000000+end_time.tv_usec) - (start_time.tv_sec*1000000+start_time.tv_usec);

    // Reverse bandwidth intensive stuff goes here

    gettimeofday(&end_time, NULL);
    long bw2_time = (end_time.tv_sec*1000000+end_time.tv_usec) - (start_time.tv_sec*1000000+start_time.tv_usec);


    // CPU intensive stuff goes here
#if DEBUG
    printf("cpu part start\n");
    for (int i=0; i<res_length; ++i)
    {
        printf("%d ", result[i]);
    }
    printf("\n");
#endif
    if (letter_index > result[res_length - 1])
    {
        fprintf(stderr, "Logical index out of bounds (last index: %u).\n", result[res_length - 1]);
	CUDA_CHECK(hipFree(result_gpu));
    	CUDA_CHECK(hipFree(seq_gpu));
	exit(-1);
    }

    if (!letter_index)
    {
        fprintf(stderr, "Please used 1-based indexing (for whatever reason).\n");
	CUDA_CHECK(hipFree(result_gpu));
	CUDA_CHECK(hipFree(seq_gpu));
	exit(-1);
    }

    /**
     * Okay, now we have a buffer which contains a tree of sums, looking
     * something like this:
     *                  _
     *        4          |
     *      /   \        |
     *    3       1      |- result buffer
     *   / \     / \     |
     *  2   1   1   0   _|
     * / \ / \ / \ / \
     * A G - T C - - -  --- sequence buffer
     *
     * (actually, it looks more like 2 1 1 2 3 3 6)
     *
     * Now, we walk through it from the top. Let's assume we're looking for the
     * logical index 2. We'll compare it to 4: Of course, it's smaller (that was
     * the assertition right before this comment), else, we'd be out of bounds.
     * No we're comparing it with the left 3 in the next level. It's smaller,
     * therefore, this subtree is correct and we move on to the next level.
     * There, we compare it to the left 2. 2 is greater/equal to 2, therefore,
     * this is _not_ the right subtree, we have to go to the other one (the one
     * to the right, below the 1). We subtract the 2 from the left subtree,
     * therefore our new "local" index is 0 (we're looking for the nucleobase at
     * index 0 in the subtree below the 1). Now, at the sequence level, there
     * are always just two possibilities. Either, the local index is 0 or it is
     * 1. If it's 1, this will always mean the right nucleobase, since 1 means
     * to skip one. The only one to skip is the left one, therefore, the right
     * one is the one we're looking for. If the local index is 0, this refers to
     * the first nucleobase, which may be either the left or the right,
     * depending on whether the left one is actually a nucleobase.
     *
     * In this case, the local index is 0. Since the left nucleobase is not
     * really one (it is '-'), the right one is the one we're looking for; its
     * index in the sequence buffer is 3.
     *
     * The reference implementation seems to go total hazels, since it
     * apparently uses 1-based indexing. Logical index 2 would refer to G for
     * it, therefore it returns 2 (which is the 1-based index of G in the
     * sequence buffer). I can't see it from the code, but that is what the
     * result is.
     *
     *
     * For another BASE than 2, it looks like this (BASE 4):
     *
     *                9
     *        //             \\
     *    3       1       3       2
     *  // \\   // \\   // \\   // \\
     * A G - T C - - - C - T T A G - -
     *
     * Let's assume, we're looking for index 5. Compare it to 9, it's smaller,
     * so this is the tree we're looking for. Then compare it to all subtrees:
     * 5 is greater than 3, so go right and subtract 3 from 5. 2 is greater than
     * 1, so go right and subtract 1 from 2. 1 then is smaller than 3, so the
     * third subtree from the left is the one we want to enter now. The index 1
     * here refers to the first T, therefore, it is globally the second T in the
     * sequence.
     */

    // "Current level subtree starting index"; index of the first subtree sum in
    // the current level (we skip level 0, i.e., the complete tree)
    clstsi = res_length - 1 - BASE;
    // "Current level subtree count"; number of subtrees in the current level
    clstc = BASE;
    // "Current level subtree offset"; index difference of the actual set of
    // subtrees we're using from the first one in the current level
    clsto = 0;
    // Turn 1-based index into 0-based
    local_index = letter_index - 1;

    for (;;)
    {
        int subtree;

        // "First subtree index", index of the first subtree we're supposed to
        // examine
        unsigned fsti = clstsi + clsto * BASE;

        // We could add a condition (subtree < BASE) to this loop, but this loop
        // has to be left before this condition is false anyway (otherwise,
        // something is very wrong).
        for (subtree = 0; local_index >= result[fsti + subtree]; subtree++)
            local_index -= result[fsti + subtree];

        // And we'll check it here anyway (#ifdef NDEBUG).
        assert(subtree < BASE);


        clsto = clsto * BASE + subtree;

        // If clstsi is 0, we were at the beginning of the result buffer and are
        // therefore finished
        if (!clstsi)
            break;

        clstc *= BASE;
        clstsi -= clstc;
    }

    // Now we need to go to the sequence level which requires an extra step.
    unsigned index;
    for (index = clsto * BASE; local_index; index++)
        if (sequence[index] != '-')
            local_index--;

    /*** END OF ROCKET SCIENCE LEVEL RUNTIME-TIME INTENSIVE STUFF ***/

    //replace with gettimeofday for CUDA
    gettimeofday(&end_time, NULL);
    delta_time = (end_time.tv_sec*1000000+end_time.tv_usec) - (start_time.tv_sec*1000000+start_time.tv_usec);
    printf("%li us elapsed total\n", delta_time);
    printf(" - %li us on bandwidth forth\n", bw1_time);
    printf(" - %li us on GPU\n", gpu_time - bw1_time);
    printf(" - %li us on bandwidth back\n", bw2_time - gpu_time);
    printf(" - %li us on CPU\n", delta_time - bw2_time);

    printf("Index for %u: %u\n", letter_index, index);
    printf("cnt = %u (index + 1)\n", index + 1);


    //free resource
    CUDA_CHECK(hipFree(result_gpu));
    CUDA_CHECK(hipFree(seq_gpu));

    return 0;
}
